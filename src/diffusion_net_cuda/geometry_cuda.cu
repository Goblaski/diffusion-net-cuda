#include "hip/hip_runtime.h"
#include <vector>

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "macros.hpp"
#include "geometry.hpp"
#include "geometry_cuda.hcu"

hipEvent_t startTimer() {
    hipEvent_t start;
    hipEventCreate(&start);
    hipEventRecord(start);

    return start;
}

void stopTimer(hipEvent_t start, std::string kernelName) {
    hipEvent_t stop;
    hipEventCreate(&stop);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << kernelName << ": " << milliseconds << "ms" << std::endl;
}

void checkError(std::string kernelName) {
    std::string error = hipGetErrorString(hipPeekAtLastError());
    std::cout << kernelName << ": " << error << std::endl;
    error = hipGetErrorString(hipDeviceSynchronize());
    std::cout << kernelName << ": " << error << std::endl;
}

void assign_vert_edge_outgoing_cuda(
    torch::Tensor edges,
    torch::Tensor vert_edge_outgoing,
    torch::Tensor vert_edge_outgoing_count
) {
    int blocks = ceilf(edges.size(1) / (float)BLOCK_SIZE);    

    #if DEBUG
        hipEvent_t start = startTimer();
    #endif

    kernel::assign_vert_edge_outgoing_cuda_kernel<<<blocks, BLOCK_SIZE>>>(
        edges.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        vert_edge_outgoing.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        vert_edge_outgoing_count.packed_accessor32<int,1,torch::RestrictPtrTraits>()
    );

    #if DEBUG
        stopTimer(start, "assign_vert_edge_outgoing_cuda");

        checkError("assign_vert_edge_outgoing_cuda");
    #endif
}

void vertices_mapping_lookup_cuda(
    const int num_vertices_new,
    const int num_vertices_old,
    torch::Tensor vertices_new,
    torch::Tensor vertices_old,
    torch::Tensor mapping_new_to_old
) {
    int blocks = ceilf(num_vertices_new / (float)BLOCK_SIZE);    

    #if DEBUG
        hipEvent_t start = startTimer();
    #endif

    kernel::vertices_mapping_lookup_cuda_kernel<<<blocks, BLOCK_SIZE>>>(
        num_vertices_new,
        num_vertices_old,
        vertices_new.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        vertices_old.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        mapping_new_to_old.packed_accessor32<int,1,torch::RestrictPtrTraits>()
    );

    #if DEBUG
        stopTimer(start, "vertices_mapping_lookup_cuda");

        checkError("vertices_mapping_lookup_cuda");
    #endif
}

void vertices_mapping_close_cuda(
    const int num_vertices_lookup,
    const int num_vertices_marker,
    const float max_distance_squared,
    torch::Tensor vertices_lookup,
    torch::Tensor vertices_marker,
    torch::Tensor mapped_close_vertices
) {
    int blocks = ceilf(num_vertices_lookup / (float)BLOCK_SIZE);    

    #if DEBUG
        hipEvent_t start = startTimer();
    #endif

    kernel::vertices_mapping_close_cuda_kernel<<<blocks, BLOCK_SIZE>>>(
        num_vertices_lookup,
        num_vertices_marker,
        max_distance_squared,
        vertices_lookup.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        vertices_marker.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        mapped_close_vertices.packed_accessor32<int,1,torch::RestrictPtrTraits>()
    );

    #if DEBUG
        stopTimer(start, "vertices_mapping_close_cuda");

        checkError("vertices_mapping_close_cuda");
    #endif
}


void get_minv_matrix_cuda(
    const int num_normals,
    const torch::Tensor normals,
    const torch::Tensor k,
    torch::Tensor minv
) {
    int blocks = ceilf(num_normals / (float)BLOCK_SIZE);    

    #if DEBUG
        hipEvent_t start = startTimer();
    #endif

    kernel::get_minv_matrix_cuda_kernel<<<blocks, BLOCK_SIZE>>>(
        num_normals,
        normals.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        k.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        minv.packed_accessor32<float,3,torch::RestrictPtrTraits>()
    );

    #if DEBUG
        stopTimer(start, "get_minv_matrix_cuda");

        checkError("get_minv_matrix_cuda");
    #endif
}



void build_grad_compressed_cuda(
        const int num_vertices,
        const int max_nhood,
        torch::Tensor edges,
        torch::Tensor edge_tangent_vectors,
        torch::Tensor vert_edge_outgoing,
        torch::Tensor vert_edge_outgoing_count,
        torch::Tensor row_inds,
        torch::Tensor col_inds,
        torch::Tensor data_vals_real,
        torch::Tensor data_vals_imag,
        const float eps_reg,
        const float w_e
    ) {

    int blocks = ceilf(num_vertices / (float)BLOCK_SIZE);    

    #if DEBUG
        hipEvent_t start = startTimer();
    #endif
    
    kernel::build_grad_compressed_cuda_kernel<<<blocks, BLOCK_SIZE>>>(
        num_vertices,
        edges.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        edge_tangent_vectors.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        vert_edge_outgoing.packed_accessor32<int,2,torch::RestrictPtrTraits>(),
        vert_edge_outgoing_count.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
        row_inds.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
        col_inds.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
        data_vals_real.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
        data_vals_imag.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
        eps_reg,
        w_e
    );

    #if DEBUG
        stopTimer(start, "build_grad_compressed_cuda");

        checkError("build_grad_compressed_cuda");
    #endif
   
    }